//headers
#include <hip/hip_runtime.h>

//cuda kernel definition
__global__ void sinewave_kernel(float4 *pos, unsigned int width, unsigned int height, float timer)
{   
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    float u, v, w;
    const float frequency = 4.0f;

    u = x / (float)width;
    v = y / (float)height;

    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    w = sinf(u * frequency + timer) * cosf(v * frequency + timer) * 0.5f;

    pos[y * width + x] = make_float4(u, w, v, 1.0f);
}

void launch_cuda_kernel(float4 *ppos, unsigned int width, unsigned int height, float timer)
{
    dim3 block = dim3(8, 8, 8);
    dim3 grid = dim3(width/block.x, height/block.y, 1);

    sinewave_kernel<<<grid, block>>>(ppos, width, height, timer);
}
